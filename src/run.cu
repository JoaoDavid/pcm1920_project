#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

extern "C" { 
    #include "../include/tree_generator.h"
    #include "../include/node.h"
    #include "../include/stack.h"
    #include "../include/dataset_parser.h"
}

#define NUM_TREES 2
#define NUM_GENERATIONS 2
/*#define NUM_TREES 4000
#define NUM_GENERATIONS 3000*/
void process_tree(const float *dataset, int num_vars, int row_index, struct stack_t* stack, struct node_t* node);
void process_tree_aux(const float *dataset, int num_vars, int row_index, struct stack_t* stack, struct node_t* node);


#define DATASET(row, column) dataset[row * num_vars + column]
#define populationULT(tree, row) population[tree * num_rows + row]

void process_tree(const float *dataset, int num_vars, int row_index, struct stack_t* stack, struct node_t* node) { 
    if (node == NULL) {
        return; 
    }
    // then recur on right subtree 
    process_tree(dataset, num_vars, row_index, stack, node->right);
    // first recur on left subtree 
    process_tree(dataset, num_vars, row_index, stack, node->left);       
    // now deal with the node 
    process_tree_aux(dataset, num_vars, row_index, stack, node);
}

void process_tree_aux(const float *dataset, int num_vars, int row_index, struct stack_t* stack, struct node_t* node) {
    switch(node->c_type){
        case CT_LITERAL:{
            //printf("%d ",node->content.literal);
            push(stack, (float)node->content.literal);
            break;
        }
        case CT_DATASET_VAR:{
            //push(stack, dataset[row_index][node->content.index_in_dataset]);
            float value = DATASET(row_index, node->content.index_in_dataset);
            //printf("value in dataset %f\n", value);
            //printf("%f ",value); 
            push(stack, value);            
            break;
        }
        case CT_OPERATOR:{
            switch(node->content.operator_code){
                case OP_TIMES:{
                    //printf("* ");
                    float result = pop(stack) * pop(stack);                    
                    push(stack, result);
                    break;
                }
                case OP_PLUS:{
                    //printf("+ ");
                    float result = pop(stack) + pop(stack);
                    push(stack, result);
                    break;
                }
                case OP_MINUS:{
                    //printf("- ");
                    float result = pop(stack) - pop(stack);
                    push(stack, result);
                    break;
                }
                case OP_DIVIDE:{
                    //printf("/ ");
                    float dividend = pop(stack);
                    float divisor = pop(stack);
                    if (divisor == 0) {
                        push(stack, 0);
                    } else {
                        push(stack, dividend/divisor);
                    }                    
                    break;
                }
            }
            break;
        }
    }
}

__device__ float sigmoid(float x) {
    return 1 / (1 + exp(-x));
}

float sigmoid_cpu(float x) {
    return 1 / (1 + exp(-x));
}


__global__ void gpu_generations(int *dev_matrix_gen, float *dev_old_fitness, float *dev_new_fitness) {
    //extern __shared__ float shared[];//shared_curr_matrix_line
    //shared[threadIdx.x] = dev_old_fitness[threadIdx.x];
    dev_matrix_gen[threadIdx.x] = threadIdx.x; //0 * NUM_TREES + threadIdx.x = threadIdx.x
    
    int min_fitness_index = 0;
    for(int gen = 1; gen < NUM_GENERATIONS; gen++) {
        //__syncthreads();
        if (gen % 2 == 0) {
            min_fitness_index = threadIdx.x % 2;
            for(int i = min_fitness_index + 2; i < NUM_TREES; i += 2) {
                min_fitness_index = (dev_old_fitness[min_fitness_index] < dev_old_fitness[i] ? min_fitness_index : i);
            }
            dev_matrix_gen[gen * NUM_TREES + threadIdx.x] = min_fitness_index;
        } else {
            int boundary = NUM_TREES / 2;
            if (threadIdx.x < boundary) {
                min_fitness_index = 0;
                for(int i = min_fitness_index + 1; i < boundary; i++) {
                    min_fitness_index = (dev_old_fitness[min_fitness_index] < dev_old_fitness[i] ? min_fitness_index : i);
                }
                dev_matrix_gen[gen * NUM_TREES + threadIdx.x] = min_fitness_index;
            } else {
                min_fitness_index = boundary;
                for(int i = min_fitness_index + 1; i < NUM_TREES; i++) {
                    min_fitness_index = (dev_old_fitness[min_fitness_index] < dev_old_fitness[i] ? min_fitness_index : i);
                }
                dev_matrix_gen[gen * NUM_TREES + threadIdx.x] = min_fitness_index;
            }
        }
        //Calculate new fitness
        //__syncthreads();
        dev_new_fitness[threadIdx.x] = dev_old_fitness[threadIdx.x] + sigmoid(dev_old_fitness[min_fitness_index]);
        //__syncthreads();
        dev_old_fitness[threadIdx.x] = dev_new_fitness[threadIdx.x];
    }
}


__global__ void gpu_calc_init_fitness(float *dev_population, float *dev_target_values, int num_rows, float *dev_fitness) {
    extern __shared__ float shared[];
    //populationULT(tree, row) population[tree * num_rows + row] 
    float res = pow(dev_population[blockIdx.x * num_rows + threadIdx.x] - dev_target_values[threadIdx.x], 2);
    //shared[threadIdx.x] = pow(dev_population[blockIdx.x * num_rows + threadIdx.x] - dev_target_values[threadIdx.x], 2);
    shared[threadIdx.x] = res;
    //dev_population[blockIdx.x * num_rows + threadIdx.x] = res;//pode tirar-se
    //__syncthreads();
    int i = num_rows/2;
    int j = num_rows%2;
    while (i != 0) {
        if (threadIdx.x < i) {
            shared[threadIdx.x] += shared[threadIdx.x + i];
            //dev_population[blockIdx.x * num_rows + threadIdx.x] += shared[blockIdx.x * num_rows + threadIdx.x + i];//pode tirar-se
        }
        if (j != 0 && threadIdx.x == 0) {
            shared[threadIdx.x] += shared[i * 2];
            //dev_population[blockIdx.x * num_rows + threadIdx.x] += dev_population[blockIdx.x * num_rows + (i*2)];
        }
        __syncthreads();
        i /= 2;
        j = i % 2;
    }

    if(threadIdx.x == 0) {
        dev_fitness[blockIdx.x] = shared[threadIdx.x] / num_rows;
        //dev_fitness[blockIdx.x] = dev_population[blockIdx.x * num_rows + threadIdx.x] / num_rows;
    }
}

void gpu_preparation(float *population, float *target_values, int *matrix_gen, float *gpu_fitness, int target_values_size, int population_size, int matrix_gen_size, int num_rows) {
    float *dev_population;
    hipMalloc(&dev_population, population_size);
    hipMemcpy(dev_population, population, population_size, hipMemcpyHostToDevice);

    float *dev_target_values; //pointer to the location of the y's values in the gpu's memory
    hipMalloc(&dev_target_values, target_values_size);
    hipMemcpy(dev_target_values, target_values, target_values_size, hipMemcpyHostToDevice);
    
    float *dev_fitness;
    hipMalloc(&dev_fitness, NUM_TREES * sizeof(float));

    float *new_fitness = (float*) malloc(NUM_TREES * sizeof(float));
    float *dev_new_fitness;
    hipMalloc(&dev_new_fitness, NUM_TREES * sizeof(float));


    //Prints dataset content
    /*printf("---------- before PRINTING population CONTENT ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        for(int j = 0; j < num_rows; j++){
            printf("%f ", populationULT(i,j));
        }
        printf("\n");
    }*/

    gpu_calc_init_fitness<<<NUM_TREES, num_rows, sizeof(float) * num_rows>>>(dev_population, dev_target_values, num_rows, dev_fitness);
    hipMemcpy(population, dev_population, population_size, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_fitness, dev_fitness, NUM_TREES*sizeof(float), hipMemcpyDeviceToHost);

    //Prints dataset content
    /*printf("---------- after PRINTING population CONTENT ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        for(int j = 0; j < num_rows; j++){
            printf("%f ", populationULT(i,j));
        }
        printf("\n");
    }*/

    //Prints fitness
    /*printf("---------- first fitness gpu ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        printf("%f , ", gpu_fitness[i]);
    }
    printf("\n");*/
    int *dev_matrix_gen;
    hipMalloc(&dev_matrix_gen, matrix_gen_size);
    gpu_generations<<<1, NUM_TREES>>>(dev_matrix_gen, dev_fitness, dev_new_fitness);
    hipMemcpy(matrix_gen, dev_matrix_gen, matrix_gen_size, hipMemcpyDeviceToHost);
    hipMemcpy(gpu_fitness, dev_new_fitness, NUM_TREES*sizeof(float), hipMemcpyDeviceToHost);

    /*printf("---------- PRINTING MATRIX GEN ----------\n");
    for(int i = 0; i < NUM_GENERATIONS; i++) {
        for(int j = 0; j < NUM_TREES; j++){
            printf("%d ", matrix_gen[i * NUM_TREES + j]);
        }
        printf("\n");
    }*/

}



void cpu_seq_version(float *population, float *target_values, int *cpu_matrix_gen, float *old_fitness, int num_rows) {
    //float *old_fitness = (float*) malloc(NUM_TREES * sizeof(float));
    float *new_fitness = (float*) malloc(NUM_TREES * sizeof(float));
    float *aux;

    for(int i = 0; i < NUM_TREES; i++) {
        float curr = 0;
        for(int j = 0; j < num_rows; j++){
            curr += pow(population[i * num_rows + j] - target_values[j],2);
        }
        old_fitness[i] = curr / num_rows;
    }
    /*printf("---------- first fitness cpu ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        printf("%f , ", old_fitness[i]);
    }
    printf("\n");*/
    // second part of the algorithm
    for(int i = 0; i < NUM_TREES; i++) { //gen = 0
        cpu_matrix_gen[i] = i;
    }
    for(int gen = 1; gen < NUM_GENERATIONS; gen++) {
        if (gen % 2 == 0) {
            int min_fitness_index_even = 0;
            int min_fitness_index_odd = 1;
            for(int i = 2; i < NUM_TREES; i++) {
                if (i % 2 == 0) {
                    min_fitness_index_even = (old_fitness[min_fitness_index_even] < old_fitness[i] ? min_fitness_index_even : i);
                } else {
                    min_fitness_index_odd = (old_fitness[min_fitness_index_odd] < old_fitness[i] ? min_fitness_index_odd : i);
                }
            }
            for(int i = 0; i < NUM_TREES; i++) {
                if (i % 2 == 0) {
                    cpu_matrix_gen[gen * NUM_TREES + i] = min_fitness_index_even;
                    new_fitness[i] = old_fitness[i] + sigmoid_cpu(old_fitness[min_fitness_index_even]);
                } else {
                    cpu_matrix_gen[gen * NUM_TREES + i] = min_fitness_index_odd;
                    new_fitness[i] = old_fitness[i] + sigmoid_cpu(old_fitness[min_fitness_index_odd]);
                }
            }
        } else {
            int boundary = NUM_TREES / 2;
            int min_fitness_index_first_half = 0;
            int min_fitness_index_second_half = boundary;
            for(int i = 1; i < NUM_TREES; i++) {
                if (i < boundary) {
                    min_fitness_index_first_half = (old_fitness[min_fitness_index_first_half] < old_fitness[i] ? min_fitness_index_first_half : i);
                } else {
                    min_fitness_index_second_half = (old_fitness[min_fitness_index_second_half] < old_fitness[i] ? min_fitness_index_second_half : i);
                }
            }
            for(int i = 0; i < NUM_TREES; i++) {
                if (i < boundary) {
                    cpu_matrix_gen[gen * NUM_TREES + i] = min_fitness_index_first_half;
                    new_fitness[i] = old_fitness[i] + sigmoid_cpu(old_fitness[min_fitness_index_first_half]);
                } else {
                    cpu_matrix_gen[gen * NUM_TREES + i] = min_fitness_index_second_half;
                    new_fitness[i] = old_fitness[i] + sigmoid_cpu(old_fitness[min_fitness_index_second_half]);
                }
            }
        }
        /*aux = old_fitness;
        old_fitness = new_fitness;
        new_fitness = aux;  */
        //printf("insnide gen\n");
        for(int i = 0; i < NUM_TREES; i++) {
            //printf("%f ", new_fitness[i]);
            old_fitness[i] = new_fitness[i];
        }//printf("\n");
    }
    //memcpy(fitness, new_fitness, NUM_TREES*sizeof(float));
    //memcpy(fitness, old_fitness, NUM_TREES);
    /*printf("---------- PRINTING MATRIX GEN  cpu----------\n");
    for(int i = 0; i < NUM_GENERATIONS; i++) {
        for(int j = 0; j < NUM_TREES; j++){
            printf("%d ", cpu_matrix_gen[i * NUM_TREES + j]);
        }
        printf("\n");
    }printf("\n"); */
}


int main(int argc, char *argv[]) {
    #define TIMER_START() gettimeofday(&tv1, NULL)
    #define TIMER_STOP()                                                           \
    gettimeofday(&tv2, NULL);                                                    \
    timersub(&tv2, &tv1, &tv);                                                   \
    time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

    struct timeval tv1, tv2, tv;
    float time_delta;


    //srand(time(NULL));
    //Parsing dataset file, and adding its values to the dataset array
    int num_columns = parse_file_columns(argv[1]); //x0,x1,x2,x3,...,xn and y
    int num_rows = parse_file_rows(argv[1]);
    int num_vars = num_columns - 1; //excluding y
    float* dataset = (float*) malloc((num_columns-1)*num_rows*sizeof(float));
    int target_values_size = num_rows*sizeof(float);
    float* target_values = (float*) malloc(target_values_size);
    parse_file_data(argv[1],dataset,target_values,num_columns,num_rows);
    printf("Dataset rows: %d\n",num_rows);
    printf("Dataset columns: %d\n",num_columns);    

    //Generating trees and processing the results with the dataset array
    struct node_t *trees[NUM_TREES];
    struct stack_t* stack = create_stack();
    float total_size = 0;
    int population_size = NUM_TREES*num_rows*sizeof(float);
    float* population = (float*) malloc(population_size);
    for(int i = 0; i < NUM_TREES; i++) {
        trees[i] = generate_tree(num_vars);
        //print_tree_rpn(trees[i]); printf("\n");
        total_size += tree_size(trees[i]);
        for(int j = 0; j < num_rows; j++){
            process_tree(dataset,num_vars,j,stack,trees[i]);
            populationULT(i,j) = pop(stack);
            //printf("Result (tree:%d|row:%d) %f\n", i, j, populationULT(i,j));
            clean_stack(stack);
        }
    }

    //Prints dataset content
    /*printf("---------- PRINTING DATASET CONTENT ----------\n");
    for(int i = 0; i < num_rows; i++) {
        for(int j = 0; j < num_vars; j++){
            printf("%f ", DATASET(i,j));
        }
        printf("%f \n", target_values[i]);
    }*/

    int matrix_gen_size = NUM_TREES * NUM_GENERATIONS * sizeof(int);
    int *cpu_matrix_gen = (int*) malloc(matrix_gen_size);
    int *gpu_matrix_gen = (int*) malloc(matrix_gen_size);
    float *cpu_fitness = (float*) malloc(NUM_TREES * sizeof(float));
    float *gpu_fitness = (float*) malloc(NUM_TREES * sizeof(float));

    fprintf(stderr, "running on cpu...  ");
    TIMER_START();
    cpu_seq_version(population, target_values, cpu_matrix_gen, cpu_fitness, num_rows);
    TIMER_STOP();
    fprintf(stderr, "%f secs\n", time_delta);

    /*printf("---------- final fitness cpu ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        printf("%f , ", cpu_fitness[i]);
    }
    printf("\n");*/

    fprintf(stderr, "running on gpu...  ");
    TIMER_START();
    gpu_preparation(population, target_values, gpu_matrix_gen, gpu_fitness, target_values_size, population_size, matrix_gen_size, num_rows);
    TIMER_STOP();
    fprintf(stderr, "%f secs\n", time_delta);

    /*printf("---------- final fitness gpu ----------\n");
    for(int i = 0; i < NUM_TREES; i++) {
        printf("%f , ", gpu_fitness[i]);
    }
    printf("\n");*/

    free(dataset);
    free(target_values);
    free(population);
    destroy_stack(stack);
    
    float average = (float)(total_size/NUM_TREES);
    printf("average tree size is %lf\n", average);
    for(int i = 0; i < NUM_TREES; i++) {
        node_destroy(trees[i]);
    }    

    float espilon = 0.000001;
    for(int i = 0; i < NUM_TREES; i++) {
        if (cpu_fitness[i] - gpu_fitness[i] >= espilon) {
            printf("cpu and gpu final fitness arrays are different, values %f ; %f - FAIL!\n",cpu_fitness[i],gpu_fitness[i]);
            break;
        }        
    }
    if (memcmp(cpu_matrix_gen, gpu_matrix_gen, matrix_gen_size) != 0) {
        fprintf(stderr, "final matrix FAIL!- FAIL!- FAIL!- FAIL!\n");
    } else {
        printf("cpu and gpu matrixes are equal - OK\n");
    }

}